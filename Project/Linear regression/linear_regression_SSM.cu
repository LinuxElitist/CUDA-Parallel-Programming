#include "hip/hip_runtime.h"
/*This code implements Linear Regression using Static Shared Memory*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>
struct timeval begin, End;
unsigned long long time_spent;

//global variables
#define NUM_ELEMENTS 262144 //must be k*BLOCK_SIZE
#define NUM_BLOCK 32
#define BLOCK_SIZE 1024 //this will make sure everytime only one block in SM

//these two values should be the 2*#SM in GPU, streaming multiprocessors
//we have change to schedule all block in one stage, we have no necessity to put them in two. 
#define CLEAN_SCOPE 16
#define COLLECT_SCOPE 16
//design the case: one SM has one block at one time (1024 threads)
//one block rewrite all the SSM to 0. 
#define SM_EXPAND 6 //this is 48KB/(1K*sizeof(double)) = 12

/* GPU code */
__global__ void clean_ssm(){
        __shared__ double ssm[1024*SM_EXPAND]; //right, BLOCK_SIZE*SM_EXPAND+1 will report error. 
        int id = threadIdx.x;

        int i;
        for(i = 0; i<SM_EXPAND; i++){
                ssm[id + BLOCK_SIZE*i] = 0.0;
        }
        //data[blockIdx.x*BLOCK_SIZE + id] = ssm[id]; 
}

__global__ void linear_regression_ssm(float *data_x, float *data_y, int n){
        __shared__ double sum[5 + 4*BLOCK_SIZE]; //sum[4*BLOCK_SIZE+4] is flag, sum[4*BLOCK_SIZE - +3] is value

        unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned tid = threadIdx.x;
        unsigned gridsize = blockDim.x * gridDim.x;

        double local_xy = 0;
        double local_xx = 0;
        double local_x = 0;
        double local_y = 0;

        while(id < n){ //need #threads is k*blocksize
                local_xy += data_x[id]*data_y[id];
                local_xx += data_x[id]*data_x[id];
                local_x += data_x[id];
                local_y += data_y[id];
                id += gridsize;
        }
        sum[tid] = local_xy;
        sum[1*BLOCK_SIZE+tid] = local_xx;
        sum[2*BLOCK_SIZE+tid] = local_x;
        sum[3*BLOCK_SIZE+tid] = local_y;
        __syncthreads();

        for (unsigned stride = (blockDim.x/2); stride >= 1;  stride >>= 1) {
                if (tid < stride){ //need #threads is k*blocksize
                        sum[tid] += sum[tid + stride];
                        sum[1*BLOCK_SIZE+tid] += sum[1*BLOCK_SIZE+tid + stride];
                        sum[2*BLOCK_SIZE+tid] += sum[2*BLOCK_SIZE+tid + stride];
                        sum[3*BLOCK_SIZE+tid] += sum[3*BLOCK_SIZE+tid + stride];
                }
                __syncthreads();
        }

        if(tid == 0){
                sum[4*BLOCK_SIZE+4] = 1;
                sum[4*BLOCK_SIZE] += sum[0];
                sum[4*BLOCK_SIZE+1] += sum[1*BLOCK_SIZE];
                sum[4*BLOCK_SIZE+2] += sum[2*BLOCK_SIZE];
                sum[4*BLOCK_SIZE+3] += sum[3*BLOCK_SIZE];
        }
}

__global__ void collect_ssm(double *result){
        __shared__ double sum[5 + 4*BLOCK_SIZE]; //
        if(threadIdx.x == 0){
                if(sum[4*BLOCK_SIZE+4] == 1){
                        result[blockIdx.x*4] = sum[4*BLOCK_SIZE];
                        result[blockIdx.x*4+1] = sum[4*BLOCK_SIZE+1];
                        result[blockIdx.x*4+2] = sum[4*BLOCK_SIZE+2];
                        result[blockIdx.x*4+3] = sum[4*BLOCK_SIZE+3];
                        sum[4*BLOCK_SIZE+4] = 0;
                }
        }
}

/* CPU code */
int main(int argc, char **argv){
        float ans_b1, ans_b0;
        if(argc != 3){
                printf("error argument input, should be: $xx_regression b0 b1, 2 float/int number \n");
                return 0;
        }
        ans_b0 = atof(argv[1]);
        ans_b1 = atof(argv[2]);

        int i;
        float *h_data_x, *h_data_y; //host data x y
        //they are: sum(xi * yi), sum(xi^2), sum(xi), sum(yi)
        double h_A, h_B, h_C, h_D;
        //they are: mean(x), mean(y)
        double temp_A, temp_B;
        double h_b1, h_b0, d_b1, d_b0;


        srand(time(NULL));
        h_A = 0; h_B = 0; h_C = 0; h_D = 0;

        h_data_x = (float *)malloc(NUM_ELEMENTS * sizeof(float));
        h_data_y = (float *)malloc(NUM_ELEMENTS * sizeof(float));
//////
printf("\nHost data: \n");
        for(i = 0; i < NUM_ELEMENTS; i++){
                h_data_x[i] = (rand()%1000)/10.0; //x from 0-100
                h_data_y[i] = ans_b0 + ans_b1 * h_data_x[i] + (rand()%80 - 40)/10.0; //y will have +-4 change

                h_A += h_data_x[i]*h_data_y[i];
                h_B += h_data_x[i]*h_data_x[i];
                h_C += h_data_x[i];
                h_D += h_data_y[i];
//////
//printf("[%d,%d] ", i, h_data[i]); 
        }
//////
printf("\n");

        temp_A = h_C/NUM_ELEMENTS; //mean x
        temp_B = h_D/NUM_ELEMENTS; //mean y
        h_b1 = (h_A - NUM_ELEMENTS*temp_A*temp_B)/(h_B - NUM_ELEMENTS*temp_A*temp_A);
        h_b0 = temp_B - h_b1 * temp_A;


        double result[4*COLLECT_SCOPE]; //for device h_A - h_D
        for(i = 0; i<(4*COLLECT_SCOPE); i++){
                result[i] = 0;
        }

gettimeofday(&begin, NULL);

        float *d_data_x, *d_data_y; //device data
        double *d_result;
        hipMalloc((void**)&d_data_x, NUM_ELEMENTS*sizeof(float));
        hipMalloc((void**)&d_data_y, NUM_ELEMENTS*sizeof(float));
        hipMemcpy(d_data_x, h_data_x, NUM_ELEMENTS*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_data_y, h_data_y, NUM_ELEMENTS*sizeof(float), hipMemcpyHostToDevice);
        hipMalloc((void**)&d_result, 4*COLLECT_SCOPE*sizeof(double));
        hipMemcpy(d_result, result, 4*COLLECT_SCOPE*sizeof(double), hipMemcpyHostToDevice);
        dim3 dim_grid(NUM_BLOCK, 1, 1);
        dim3 dim_block(BLOCK_SIZE, 1, 1);

        clean_ssm<<<CLEAN_SCOPE, 1024>>>();
        hipDeviceSynchronize();
        linear_regression_ssm<<<dim_grid, dim_block>>>(d_data_x, d_data_y, NUM_ELEMENTS); //128 1024
        hipDeviceSynchronize();
        collect_ssm<<<COLLECT_SCOPE, 1024>>>(d_result);
        hipDeviceSynchronize();

        hipMemcpy(result, d_result, 4*COLLECT_SCOPE*sizeof(double), hipMemcpyDeviceToHost);
        hipFree(d_data_x);
        hipFree(d_data_y);
        hipFree(d_result);

gettimeofday(&end, NULL);
time_spent = 1000000 * (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) ;
printf("linear_regression with SSM computation cost time: %llu us \n", time_spent);

        h_A = 0; h_B = 0; h_C = 0; h_D = 0;
        for(i = 0; i< COLLECT_SCOPE; i++){
                h_A += result[i*4];
                h_B += result[i*4 + 1];
                h_C += result[i*4 + 2];
                h_D += result[i*4 + 3];
        }

        temp_A = h_C/NUM_ELEMENTS; //mean x
        temp_B = h_D/NUM_ELEMENTS; //mean y
        d_b1 = (h_A - NUM_ELEMENTS*temp_A*temp_B)/(h_B - NUM_ELEMENTS*temp_A*temp_A);
        d_b0 = temp_B - d_b1 * temp_A;

        printf("input: b0: %f, b1: %f \nhost: b0: %f, b1: %f \ndevice: b0: %f, b1: %f \n",
                ans_b0, ans_b1, h_b0, h_b1, d_b0, d_b1);

        free(h_data_x);
        free(h_data_y);
        return 0;
}


