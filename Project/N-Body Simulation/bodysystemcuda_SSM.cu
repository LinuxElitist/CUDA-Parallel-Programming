#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_runtime_api.h>
#include <math.h>

#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// CUDA standard includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "bodysystem.h"

__constant__ float softeningSquared;
__constant__ double softeningSquared_fp64;

hipError_t setSofteningSquared(float softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared),
                              &softeningSq,
                              sizeof(float), 0,
                              hipMemcpyHostToDevice);
}

hipError_t setSofteningSquared(double softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared_fp64),
                              &softeningSq,
                              sizeof(double), 0,
                              hipMemcpyHostToDevice);
}

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<typename T>
__device__ T rsqrt_T(T x)
{
    return rsqrt(x);
}

template<>
__device__ float rsqrt_T<float>(float x)
{
    return rsqrtf(x);
}

template<>
__device__ double rsqrt_T<double>(double x)
{
    return rsqrt(x);
}


// Macros to simplify shared memory addressing
#define SX(i) sharedPos[i+blockDim.x*threadIdx.y]
// This macro is only used when multithreadBodies is true (below)
#define SX_SUM(i,j) sharedPos[i+blockDim.x*j]

template <typename T>
__device__ T getSofteningSquared()
{
    return softeningSquared;
}
template <>
__device__ double getSofteningSquared<double>()
{
    return softeningSquared_fp64;
}

template <typename T>
struct DeviceData
{
    T *dPos[2]; // mapped host pointers
    T *dVel;
    hipEvent_t  event;
    unsigned int offset;
    unsigned int numBodies;
};


/* SSM design include and functions START */
#include "cutil_ssm_config.h"

__global__ void SSM_CLEAN(){
        __shared__ float ssm[SSM_CLEAN_BLOCK_SIZE*SSM_CLEAN_EXPAND];
        //__shared__ double ssm[SSM_CLEAN_BLOCK_SIZE*SSM_CLEAN_EXPAND];
        int id = threadIdx.x;

        int i;
        for(i = 0; i<SSM_CLEAN_EXPAND; i++){
                ssm[id + SSM_CLEAN_BLOCK_SIZE*i] = 0.0;
        }

		//if (threadIdx.x == 0) printf("working on clean SSM initial! \n");
}

//this code doesn't need SSM_COLLECT(argument)
//but, why do not put the data into constant memory? 

/* SSM design include and functions END */




template <typename T>
__device__ typename vec3<T>::Type
bodyBodyInteraction(typename vec3<T>::Type ai,
                    typename vec4<T>::Type bi,
                    typename vec4<T>::Type bj)
{
    typename vec3<T>::Type r;

    // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    T distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
    distSqr += getSofteningSquared<T>();

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    T invDist = rsqrt_T(distSqr);
    T invDistCube =  invDist * invDist * invDist;

    // s = m_j * invDistCube [1 FLOP]
    T s = bj.w * invDistCube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;

    return ai;
}

template <typename T>
__device__ typename vec3<T>::Type
computeBodyAccel(typename vec4<T>::Type bodyPos,
                 typename vec4<T>::Type *positions,
                 int numTiles)
{
	__shared__ typename vec4<T>::Type SSM[SSM_CLEAN_BLOCK_SIZE*SSM_CLEAN_EXPAND/8];
	//size if not 4*float !!
    //typename vec4<T>::Type *sharedPos = SharedMemory<typename vec4<T>::Type>();
	typename vec4<T>::Type *sharedPos = SSM;

    typename vec3<T>::Type acc = {0.0f, 0.0f, 0.0f};

    for (int tile = 0; tile < numTiles; tile++)
    {
	//////
	//load from global memory, for position
//////
//SSM design
	//if (threadIdx.x == 0) printf("working on calculation! block id: %d \n", blockIdx.x);

	//original
	//sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

	if(sharedPos[numTiles * blockDim.x].w == 0){
        	sharedPos[tile * blockDim.x + threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

			//if (threadIdx.x == 0) printf("loading data! block id: %d, %d, %f, %f, %f, %f \n", blockIdx.x, numTiles, 
			//	sharedPos[tile * blockDim.x + threadIdx.x].x, sharedPos[tile * blockDim.x + threadIdx.x].y, sharedPos[tile * blockDim.x + threadIdx.x].z, 
			//	sharedPos[tile * blockDim.x + threadIdx.x].w);

	}
	else{
		if(threadIdx.x == 0) printf("already load data! SSM reuse! \n");
	}
	//if (threadIdx.x == 0) printf("finish loading! block id: %d \n", blockIdx.x);

        __syncthreads();

	//if (threadIdx.x == 0) printf("after sync! block id: %d, %d \n", blockIdx.x, tile);

        // This is the "tile_calculation" from the GPUG3 article.
#pragma unroll 128

	//////
	//calculate to current block data (like: matrix mul: tile to itself)
	//also calculate the entire "acc"
        for (unsigned int counter = 0; counter < blockDim.x; counter++)
        {
//////
//SSM design
			//if (threadIdx.x == 0) printf("before calculation! block id: %d, sharedPos %d \n", blockIdx.x, (tile * blockDim.x + counter));

            acc = bodyBodyInteraction<T>(acc, bodyPos, sharedPos[tile * blockDim.x + counter]);
			//acc = bodyBodyInteraction<T>(acc, bodyPos, sharedPos[counter]);
        }

	//if (threadIdx.x == 0) printf("finish calculation! block id: %d \n", blockIdx.x);

        __syncthreads();
    }

//////
//SSM design, set flag
    if(sharedPos[numTiles * blockDim.x].w == 0){
	  if(threadIdx.x == 0){
	    sharedPos[numTiles * blockDim.x].w = 1; 
	  }
    }

    return acc;
}

template<typename T>
__global__ void
integrateBodies(typename vec4<T>::Type *__restrict__ newPos,
                typename vec4<T>::Type *__restrict__ oldPos,
                typename vec4<T>::Type *vel,
                unsigned int deviceOffset, unsigned int deviceNumBodies,
                float deltaTime, float damping, int numTiles)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= deviceNumBodies)
    {
        return;
    }

    typename vec4<T>::Type position = oldPos[deviceOffset + index];

    typename vec3<T>::Type accel = computeBodyAccel<T>(position,
                                                       oldPos,
                                                       numTiles);

    // acceleration = force / mass;
    // new velocity = old velocity + acceleration * deltaTime
    // note we factor out the body's mass from the equation, here and in bodyBodyInteraction
    // (because they cancel out).  Thus here force == acceleration
    typename vec4<T>::Type velocity = vel[deviceOffset + index];

    velocity.x += accel.x * deltaTime;
    velocity.y += accel.y * deltaTime;
    velocity.z += accel.z * deltaTime;

    velocity.x *= damping;
    velocity.y *= damping;
    velocity.z *= damping;

    // new position = old position + velocity * deltaTime
    position.x += velocity.x * deltaTime;
    position.y += velocity.y * deltaTime;
    position.z += velocity.z * deltaTime;

    // store new position and velocity
    newPos[deviceOffset + index] = position;
    vel[deviceOffset + index]    = velocity;
}

template <typename T>
void integrateNbodySystem(DeviceData<T> *deviceData,
                          hipGraphicsResource **pgres,
                          unsigned int currentRead,
                          float deltaTime,
                          float damping,
                          unsigned int numBodies,
                          unsigned int numDevices,
                          int blockSize,
                          bool bUsePBO)
{
    if (bUsePBO)
    {
        checkCudaErrors(cudaGraphicsResourceSetMapFlags(pgres[currentRead], cudaGraphicsMapFlagsReadOnly));
        checkCudaErrors(cudaGraphicsResourceSetMapFlags(pgres[1-currentRead], cudaGraphicsMapFlagsWriteDiscard));
        checkCudaErrors(hipGraphicsMapResources(2, pgres, 0));
        size_t bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[currentRead]), &bytes, pgres[currentRead]));
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[1-currentRead]), &bytes, pgres[1-currentRead]));
    }

    for (unsigned int dev = 0; dev != numDevices; dev++)
    {
        if (numDevices > 1)
        {
            hipSetDevice(dev);
        }

        int numBlocks = (deviceData[dev].numBodies + blockSize-1) / blockSize;
        int numTiles = (numBodies + blockSize - 1) / blockSize;
        int sharedMemSize = blockSize * 4 * sizeof(T); // 4 floats for pos

//////
//SSM design, do clean
	SSM_CLEAN<<<SSM_NUM_STREAM_MULTIPROCESSOR, SSM_CLEAN_BLOCK_SIZE>>>();

	//printf("hello, done clean \n");

        integrateBodies<T><<< numBlocks, blockSize, sharedMemSize >>>
        ((typename vec4<T>::Type *)deviceData[dev].dPos[1-currentRead],
         (typename vec4<T>::Type *)deviceData[dev].dPos[currentRead],
         (typename vec4<T>::Type *)deviceData[dev].dVel,
         deviceData[dev].offset, deviceData[dev].numBodies,
         deltaTime, damping, numTiles);

	printf("hello, done calculation \n");

        if (numDevices > 1)
        {
            checkCudaErrors(hipEventRecord(deviceData[dev].event));
            // MJH: Hack on older driver versions to force kernel launches to flush!
            hipStreamQuery(0);
        }

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    if (numDevices > 1)
    {
        for (unsigned int dev = 0; dev < numDevices; dev++)
        {
            checkCudaErrors(hipEventSynchronize(deviceData[dev].event));
        }
    }

    if (bUsePBO)
    {
        checkCudaErrors(hipGraphicsUnmapResources(2, pgres, 0));
    }
}


// Explicit specializations needed to generate code
template void integrateNbodySystem<float>(DeviceData<float> *deviceData,
                                          hipGraphicsResource **pgres,
                                          unsigned int currentRead,
                                          float deltaTime,
                                          float damping,
                                          unsigned int numBodies,
                                          unsigned int numDevices,
                                          int blockSize,
                                          bool bUsePBO);

template void integrateNbodySystem<double>(DeviceData<double> *deviceData,
                                           hipGraphicsResource **pgres,
                                           unsigned int currentRead,
                                           float deltaTime,
                                           float damping,
                                           unsigned int numBodies,
                                           unsigned int numDevices,
                                           int blockSize,
                                           bool bUsePBO);
